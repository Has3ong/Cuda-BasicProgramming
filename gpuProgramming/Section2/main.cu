#include <stdio.h>
#include <stdlib.h>

#include "kernel.cuh"

int main() {

	hipSetDevice(0);

	int N, M, K;
	N = M = 3;
	K = 1;

	double *A, *B, *C;

	A = (double *)malloc(sizeof(double) * N * M);
	B = (double *)malloc(sizeof(double) * M * K);
	C = (double *)malloc(sizeof(double) * N * K);

	for (int i = 0; i < N * M; i++) {
		A[i] = 1.;
	}
	for (int j = 0; j < M *K; j++) {
		B[j] = 1.;
	}
	for (int k = 0; k < N * K; k++) {
		C[k] = 0.;
	}

	return 0;
}