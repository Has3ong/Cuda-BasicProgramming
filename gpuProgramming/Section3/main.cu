#include <stdio.h>
#include <stdlib.h>

#include "kernel.cuh"

int main() {

	hipSetDevice(0);

	int N, M, K;
	N = M = 3;
	K = 1;

	double *A, *B, *C, *d_A, *d_B, *d_C;

	A = (double *)malloc(sizeof(double) * N * M);
	B = (double *)malloc(sizeof(double) * M * K);
	C = (double *)malloc(sizeof(double) * N * K);

	hipMalloc(&d_A, sizeof(double) * N * M);
	hipMalloc(&d_B, sizeof(double) * M * K);
	hipMalloc(&d_C, sizeof(double) * N * K);

	for (int i = 0; i < N * M; i++) {
		A[i] = 1.;
	}
	for (int j = 0; j < M *K; j++) {
		B[j] = 1.;
	}
	for (int k = 0; k < N * K; k++) {
		C[k] = 0.;
	}

	hipMemcpy(d_A, A, sizeof(double) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(double) * M * K, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(double) * N * K, hipMemcpyHostToDevice);

	int nTreads = N * K;
	Kernel  <<< 1, N * K >>> (N, M, K, A, B, C);

	hipMemcpy(C, d_C, sizeof(double) * N * K, hipMemcpyDeviceToHost);
	return 0;
}