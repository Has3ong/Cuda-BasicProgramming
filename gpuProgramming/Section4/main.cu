#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "kernel.cuh"

int main() {

	hipSetDevice(0);

	int N, M, K;
	N = M = 10000;
	K = 1;

	double *A, *B, *C, *d_A, *d_B, *d_C;

	A = (double *)malloc(sizeof(double) * N * M);
	B = (double *)malloc(sizeof(double) * M * K);
	C = (double *)malloc(sizeof(double) * N * K);

	hipMalloc(&d_A, sizeof(double) * N * M);
	hipMalloc(&d_B, sizeof(double) * M * K);
	hipMalloc(&d_C, sizeof(double) * N * K);

	for (int i = 0; i < N * M; i++) {
		A[i] = 1.;
	}
	for (int j = 0; j < M *K; j++) {
		B[j] = 1.;
	}
	for (int k = 0; k < N * K; k++) {
		C[k] = 0.;
	}

	hipMemcpy(d_A, A, sizeof(double) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(double) * M * K, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(double) * N * K, hipMemcpyHostToDevice);

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	int nThreads = (int)(devProp.maxThreadsPerBlock / 4);
	int nBlocks = 65535;

	//////////////////////////////////////////////////////////////////////////////////////////
	// Device Computing

	hipEvent_t d_begin, d_end;

	hipEventCreate(&d_begin);
	hipEventCreate(&d_end);

	hipEventRecord(d_begin);

	Kernel <<< nBlocks, nThreads > > > (N, M, K, d_A, d_B, d_C);

	hipEventRecord(d_end);


	hipMemcpy(C, d_C, sizeof(double) * N * K, hipMemcpyDeviceToHost);

	hipEventSynchronize(d_end);

	float timeValue;
	hipEventElapsedTime(&timeValue, d_begin, d_end);

	printf("The time of Device:\t %f \n", timeValue);


	//////////////////////////////////////////////////////////////////////////////////////////
	// Host Computing

	double *h_C;
	h_C = (double *)malloc(sizeof(double) * N * K);

	clock_t start = clock(), diff;
	
	for (int i = 0; i < N * K; i++) {
		int index = i;
		h_C[index] = 0.;
		getMulti(N, M, K, index, A, B, h_C);
	}
	
	diff = clock() - start;

	int milisec = diff * 1000 / CLOCKS_PER_SEC;

	printf("The time of Host:\t %d \n", milisec);

	//////////////////////////////////////////////////////////////////////////////////////////
	// Host Computing End

	printf("C[100]:\t %f \n", C[100]);
	printf("h_C[100]:\t %f \n", h_C[100]);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(A);
	free(B);
	free(C);

	return 0;
}