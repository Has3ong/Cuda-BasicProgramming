#include "hip/hip_runtime.h"
#include "kernel.cuh"

__host__ __device__
double getValue(int N, int M, int row, int col, double* List) {
	int index = row * M + col;
	return List[index];
}

__host__ __device__
double getRowIndex(int N, int M, int index) {
	return (int)(index / M);
}

__host__ __device__
int getColIndex(int N, int M, int index) {
	return (int)(index % M);
}

__host__ __device__
void getMulti(int N, int M, int K, int index, double* A, double* B, double* C) {
	C[index] = 0.;

	int row = getRowIndex(N, K, index);
	int col = getColIndex(N, K, index);

	for (int i = 0; i < M; i++) {
		double a = getValue(N, M, row, i, A);
		double b = getValue(M, K, i, col, B);

		C[index] += a * b;
	}
}

__global__
void Kernel(int N, int M, int K, double* A, double* B, double* C) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < N * K) {
		getMulti(N, M, K, id, A, B, C);
	}
}